#include "hip/hip_runtime.h"
//
// Created by zak on 2/4/25.
//

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <>


struct HashingFunct {
    double* a;
    double b;
    double w;
};

//  CUDA KERNELS

// CUDA Kernel to generate both normally distributed vector `a` and a uniform value `b`
__global__ void generateLSHParams(double* d_a, double* d_b, int n, double w, unsigned long long seed, int funcIndex) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Compute global thread index
    hiprandState state;
    
    // Initialize random seed for each function independently
    hiprand_init(seed + funcIndex, i, 0, &state);  

    int offset = funcIndex * n;  // Offset for storing a[i] in the correct hash function slot

    if (i < n) {
        d_a[offset + i] = hiprand_normal(&state);  // Generate normally distributed value for a
    }

    if (i == 0) { // Only one thread generates `b`
        d_b[funcIndex] = hiprand_uniform(&state) * w;  // Scale to range [0, w]
    }
}


// CUDA Kernel for element-wise multiplication and block-wise reduction
__global__ void multiplyElements(double* d_point, double* d_a, double* d_partialSum, int n) {
    __shared__ double sharedMem[256]; // Shared memory for block-level reduction
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    double product = (i < n) ? d_point[i] * d_a[i] : 0.0;
    sharedMem[tid] = product;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_partialSum[blockIdx.x] = sharedMem[0]; //d_partial sum contains the sum of every block stored on it's index in the vector
    }
}

// CUDA Kernel for final reduction sum
__global__ void sumPartialSums(double* d_partialSum, double* d_finalSum, int numBlocks) {
    __shared__ double sharedMem[256];
    int tid = threadIdx.x;
    
    if (tid < numBlocks) {
        sharedMem[tid] = d_partialSum[tid];
    } else {
        sharedMem[tid] = 0.0;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthread_bds();
    }

    if (tid == 0) {
        *d_finalSum = sharedMem[0];
    }
}

// CUDA Kernel to compute hashes in parallel
__global__ void computeHashes(double* d_points, int* d_hash1, int* d_hash2,
                              HashingFunct* d_hashfunctions1, HashingFunct* d_hashfunctions2,
                              int num_points, int L1, int L2, int D) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_points) {
        for (int j = 0; j < L1; j++) {
            d_hash1[i * L1 + j] = hashingComputingCUDA(d_points + i * D, d_hashfunctions1[j], D);
        }
        for (int j = 0; j < L2; j++) {
            d_hash2[i * L2 + j] = hashingComputingCUDA(d_points + i * D, d_hashfunctions2[j], D);
        }
    }
}


// 2️⃣ HOST FUNCTIONS

std::vector<HashingFunct> generateLSHParamsOnGPU(int n, double w, int numFunctions) {
    double *d_a, *d_b;  // Device memory for `a` and `b`
    
    // Allocate memory for `a` (numFunctions * n) and `b` (numFunctions)
    hipMalloc(&d_a, numFunctions * n * sizeof(double));
    hipMalloc(&d_b, numFunctions * sizeof(double));

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int funcIndex = 0; funcIndex < numFunctions; funcIndex++) {
        // Launch kernel separately for each hash function
        generateLSHParams<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, n, w, time(NULL), funcIndex); // god i swear this idea and how i fixed the problem is magnifico 
        hipDeviceSynchronize();
    }

    // Copy data back to host
    std::vector<double> h_a(numFunctions * n); // contiennt a de toutes les fonctions de hachages
    std::vector<double> h_b(numFunctions);

    hipMemcpy(h_a.data(), d_a, numFunctions * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_b.data(), d_b, numFunctions * sizeof(double), hipMemcpyDeviceToHost);

    // Fill the vector of HashingFunct
    std::vector<HashingFunct> hashFunctions(numFunctions);
    for (int i = 0; i < numFunctions; i++) { // à revoir si cette copie est nécessaire , ne serait il mieux de garder tous les a et les b dans deux vecteurs h_a et h_b !!!!!!!!!!!!!
        hashFunctions[i].a = d_a + i * n;  
        hashFunctions[i].b = h_b[i];       
        hashFunctions[i].w = w;
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);

    return hashFunctions;
}


// Hashing computation using CUDA
// prend en paramètre le point qui est un vecteur et une fonction de hashage et return le hashage qui est un entier (bucket )
__device__ int hashingComputingCUDA(const double* d_point, const HashingFunct& h, int n) {
    double sum = 0.0;

    // Compute dot product sequentially
    for (int i = 0; i < n; i++) {
        sum += d_point[i] * h.a[i];  // Each thread processes its full dot product
    }

    sum = (sum + h.b) / h.w;
    return static_cast<int>(sum);
}

// Host function to compute the final hash table
std::unordered_map<std::vector<int>, std::unordered_map<std::vector<int>, std::vector<std::vector<double>>>> finalHashCUDA(
        std::vector<std::vector<double>>& points, std::vector<HashingFunct>& hashfunctions1,
        std::vector<HashingFunct>& hashfunctions2, int L1, int L2) {

    int num_points = points.size();
    int D = points[0].size();
    int* d_hash1, * d_hash2; // vecteur ou sera stocké les hachages des points dans les 2 niveaux au niveau de la memoire global du gpu 
    double* d_points;
    HashingFunct* d_hashfunctions1, * d_hashfunctions2;

    hipMalloc(&d_hash1, num_points * L1 * sizeof(int));
    hipMalloc(&d_hash2, num_points * L2 * sizeof(int));
    hipMalloc(&d_points, num_points * D * sizeof(double));
    hipMalloc(&d_hashfunctions1, L1 * sizeof(HashingFunct));
    hipMalloc(&d_hashfunctions2, L2 * sizeof(HashingFunct));

    hipMemcpy(d_points, points.data(), num_points * D * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_hashfunctions1, hashfunctions1.data(), L1 * sizeof(HashingFunct), hipMemcpyHostToDevice);
    hipMemcpy(d_hashfunctions2, hashfunctions2.data(), L2 * sizeof(HashingFunct), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock; // vaut mieux remplacer par ceil num_points/threadsPerBlock
    // sou sl'hypothès que chaque point sera traité par un thread ( à revoir )
    computeHashes<<<blocksPerGrid, threadsPerBlock>>>(d_points, d_hash1, d_hash2, d_hashfunctions1, d_hashfunctions2, num_points, L1, L2, D);  
    hipDeviceSynchronize();

    std::vector<int> h_hash1(num_points * L1);
    std::vector<int> h_hash2(num_points * L2);
    hipMemcpy(h_hash1.data(), d_hash1, num_points * L1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_hash2.data(), d_hash2, num_points * L2 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_hash1);
    hipFree(d_hash2);
    hipFree(d_points);
    hipFree(d_hashfunctions1);
    hipFree(d_hashfunctions2);

    std::unordered_map<std::vector<int>, std::unordered_map<std::vector<int>, std::vector<std::vector<double>>>> result;
    for (int i = 0; i < num_points; i++) {
        std::vector<int> hash1(h_hash1.begin() + i * L1, h_hash1.begin() + (i + 1) * L1); // constructeur de vecteur en utilisant deux itérateurs 
        std::vector<int> hash2(h_hash2.begin() + i * L2, h_hash2.begin() + (i + 1) * L2);
        result[hash1][hash2].push_back(points[i]);
    }

    return result;
}

std::vector<std::vector<double>> loadDataFromCSV(const std::string& csvFilePath) {
    std::vector<std::vector<double>> dataPoints;
    std::ifstream file(csvFilePath);
    if (!file.is_open()) {
        throw std::runtime_error("Erreur : impossible d'ouvrir le fichier CSV");
    }

    std::string line;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;
        std::vector<double> point;

        std::getline(ss, value, ',');  // Ignorer le premier champ

        while (std::getline(ss, value, ',')) {
            point.push_back(std::stod(value));
        }

        dataPoints.push_back(point);
    }

    file.close();
    return dataPoints;
}

int main() {
    std::string csvFilePath = "../Data/fingerprints_class.csv";
    int L1 = 3, L2 = 2;
    double w1 = 10.0 , w2 = 5.0;
    
    std::vector<std::vector<double>> points = loadDataFromCSV(csvFilePath);
    int D = points[0].size();

    std::vector<HashingFunct> hashFunctions1 = generateLSHParamsOnGPU(D, w1, L1);
    std::vector<HashingFunct> hashFunctions2 = generateLSHParamsOnGPU(D, w2, L2);

    auto hashTable = finalHashCUDA(points, hashFunctions1, hashFunctions2, L1, L2);

    std::cout << "Hashing terminé !\n";

    return 0;
}