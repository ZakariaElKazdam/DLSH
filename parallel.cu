#include "hip/hip_runtime.h"
//
// Created by zak on 2/4/25.
//

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <>


struct HashingFunct {
    std::vector<double> a;
    double b;
    double w;
};

//  CUDA KERNELS

// CUDA Kernel to generate both normally distributed vector `a` and a uniform value `b`
__global__ void generateLSHParams(double* d_a, double* d_b, int n, double w, unsigned long long seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Compute global thread index
    hiprandState state;
    hiprand_init(seed, i, 0, &state); // Initialize cuRAND state

    if (i < n) {
        d_a[i] = hiprand_normal(&state); // Generate normally distributed value for a[i]
    }

    if (i == 0) { // Only thread 0 generates 'b'
        *d_b = hiprand_uniform(&state) * w; // Scale to range [0, w]
    }
}

// CUDA Kernel for element-wise multiplication and block-wise reduction
__global__ void multiplyElements(double* d_point, double* d_a, double* d_partialSum, int n) {
    __shared__ double sharedMem[256]; // Shared memory for block-level reduction
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    double product = (i < n) ? d_point[i] * d_a[i] : 0.0;
    sharedMem[tid] = product;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_partialSum[blockIdx.x] = sharedMem[0]; //d_partial sum contains the sum of every block stored on it's index in the vector
    }
}

// CUDA Kernel for final reduction sum
__global__ void sumPartialSums(double* d_partialSum, double* d_finalSum, int numBlocks) {
    __shared__ double sharedMem[256];
    int tid = threadIdx.x;
    
    if (tid < numBlocks) {
        sharedMem[tid] = d_partialSum[tid];
    } else {
        sharedMem[tid] = 0.0;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        *d_finalSum = sharedMem[0];
    }
}

// CUDA Kernel to compute hashes in parallel
__global__ void computeHashes(double* d_points, int* d_hash1, int* d_hash2,
                              HashingFunct* d_hashfunctions1, HashingFunct* d_hashfunctions2,
                              int num_points, int L1, int L2, int D) {  // D est la dimension du point dans l'espace
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_points) {
        for (int j = 0; j < L1; j++) {
            d_hash1[i * L1 + j] = hashingComputingCUDA(d_points + i * D, d_hashfunctions1[j]);
        }
        // à chaque point on applique L1 fonction de hachage et on stock dans i * L1 + j
        for (int j = 0; j < L2; j++) {
            d_hash2[i * L2 + j] = hashingComputingCUDA(d_points + i * D, d_hashfunctions2[j]);
        }
    }
}


// 2️⃣ HOST FUNCTIONS

std::vector<HashingFunct> generateLSHParamsOnGPU(int n, double w, int numFunctions) {
    HashingFunct* d_hashFunctions;
    hipMalloc(&d_hashFunctions, numFunctions * sizeof(HashingFunct));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numFunctions + threadsPerBlock - 1) / threadsPerBlock;

    generateLSHParams<<<blocksPerGrid, threadsPerBlock>>>(d_hashFunctions, n, w, numFunctions, time(NULL));
    hipDeviceSynchronize();

    std::vector<HashingFunct> hashFunctions(numFunctions);
    hipMemcpy(hashFunctions.data(), d_hashFunctions, numFunctions * sizeof(HashingFunct), hipMemcpyDeviceToHost);
    hipFree(d_hashFunctions);

    return hashFunctions;
}

// Hashing computation using CUDA
// prend en paramètre le point qui est un vecteur et une fonction de hashage et return le hashage qui est un entier (bucket )
int hashingComputingCUDA(std::vector<double>& point, HashingFunct& h) {
    int n = point.size();
    double *d_point, *d_a, *d_partialSum, *d_finalSum;
    double h_finalSum;

    // Allocate memory on GPU
    hipMalloc(&d_point, n * sizeof(double));
    hipMalloc(&d_a, n * sizeof(double));

    int numBlocks = (n + 255) / 256; // fonction ceil(n/256)
    hipMalloc(&d_partialSum, numBlocks * sizeof(double));
    hipMalloc(&d_finalSum, sizeof(double));

    // Copy data from CPU to GPU
    hipMemcpy(d_point, point.data(), n * sizeof(double), hipMemcpyHostToDevice); // point.data() est bien un pointeur , point toute seul ne marche pas , un std::vector
    hipMemcpy(d_a, h.a.data(), n * sizeof(double), hipMemcpyHostToDevice);

    multiplyElements<<<numBlocks, 256>>>(d_point, d_a, d_partialSum, n); // hypothese sur le hardware que le max de thread par bloc est 256 ( multiple de 32 pour le wraps )
    if (numBlocks > 1) {
        sumPartialSums<<<1, numBlocks>>>(d_partialSum, d_finalSum, numBlocks);
        hipMemcpy(&h_finalSum, d_finalSum, sizeof(double), hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(&h_finalSum, d_partialSum, sizeof(double), hipMemcpyDeviceToHost);
    }

    // Free GPU memory
    hipFree(d_point);
    hipFree(d_a);
    hipFree(d_partialSum);
    hipFree(d_finalSum);

    h_finalSum = (h_finalSum + h.b) / h.w;
    return static_cast<int>(h_finalSum);
}

// Host function to compute the final hash table
std::unordered_map<std::vector<int>, std::unordered_map<std::vector<int>, std::vector<std::vector<double>>>> finalHashCUDA(
        std::vector<std::vector<double>>& points, std::vector<HashingFunct>& hashfunctions1,
        std::vector<HashingFunct>& hashfunctions2, int L1, int L2) {

    int num_points = points.size();
    int D = points[0].size();
    int* d_hash1, * d_hash2; // vecteur ou sera stocké les hachages des points dans les 2 niveaux au niveau de la memoire global du gpu 
    double* d_points;
    HashingFunct* d_hashfunctions1, * d_hashfunctions2;

    hipMalloc(&d_hash1, num_points * L1 * sizeof(int));
    hipMalloc(&d_hash2, num_points * L2 * sizeof(int));
    hipMalloc(&d_points, num_points * D * sizeof(double));
    hipMalloc(&d_hashfunctions1, L1 * sizeof(HashingFunct));
    hipMalloc(&d_hashfunctions2, L2 * sizeof(HashingFunct));

    hipMemcpy(d_points, points.data(), num_points * D * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_hashfunctions1, hashfunctions1.data(), L1 * sizeof(HashingFunct), hipMemcpyHostToDevice);
    hipMemcpy(d_hashfunctions2, hashfunctions2.data(), L2 * sizeof(HashingFunct), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock; // vaut mieux remplacer par ceil num_points/threadsPerBlock
    // sou sl'hypothès que chaque point sera traité par un thread ( à revoir )
    computeHashes<<<blocksPerGrid, threadsPerBlock>>>(d_points, d_hash1, d_hash2, d_hashfunctions1, d_hashfunctions2, num_points, L1, L2, D);  
    hipDeviceSynchronize();

    std::vector<int> h_hash1(num_points * L1);
    std::vector<int> h_hash2(num_points * L2);
    hipMemcpy(h_hash1.data(), d_hash1, num_points * L1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_hash2.data(), d_hash2, num_points * L2 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_hash1);
    hipFree(d_hash2);
    hipFree(d_points);
    hipFree(d_hashfunctions1);
    hipFree(d_hashfunctions2);

    std::unordered_map<std::vector<int>, std::unordered_map<std::vector<int>, std::vector<std::vector<double>>>> result;
    for (int i = 0; i < num_points; i++) {
        std::vector<int> hash1(h_hash1.begin() + i * L1, h_hash1.begin() + (i + 1) * L1); // constructeur de vecteur en utilisant deux itérateurs 
        std::vector<int> hash2(h_hash2.begin() + i * L2, h_hash2.begin() + (i + 1) * L2);
        result[hash1][hash2].push_back(points[i]);
    }

    return result;
}

std::vector<std::vector<double>> loadDataFromCSV(const std::string& csvFilePath) {
    std::vector<std::vector<double>> dataPoints;
    std::ifstream file(csvFilePath);
    if (!file.is_open()) {
        throw std::runtime_error("Erreur : impossible d'ouvrir le fichier CSV");
    }

    std::string line;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;
        std::vector<double> point;

        std::getline(ss, value, ',');  // Ignorer le premier champ

        while (std::getline(ss, value, ',')) {
            point.push_back(std::stod(value));
        }

        dataPoints.push_back(point);
    }

    file.close();
    return dataPoints;
}

int main() {
    std::string csvFilePath = "../Data/fingerprints_class.csv";
    int L1 = 3, L2 = 2;
    double w1 = 10.0 , w2 = 5.0;
    
    std::vector<std::vector<double>> points = loadDataFromCSV(csvFilePath);
    int D = points[0].size();

    std::vector<HashingFunct> hashFunctions1 = generateLSHParamsOnGPU(D, w1, L1);
    std::vector<HashingFunct> hashFunctions2 = generateLSHParamsOnGPU(D, w2, L2);

    auto hashTable = finalHashCUDA(points, hashFunctions1, hashFunctions2, L1, L2);

    std::cout << "Hashing terminé !\n";

    return 0;
}